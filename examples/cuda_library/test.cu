#include "hip/hip_runtime.h"
#include <cstdio>
#include "sum.cuh"

__global__ void cuda_hello(){
    printf("Hello World from GPU %d!\n", sum(2, 3));
}

void hello_world_gpu() {
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
}
